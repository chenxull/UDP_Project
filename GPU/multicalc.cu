#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
// #include <math.h>
#include <time.h>
#include <cstring>
#include <random>

#include "hip/hip_runtime.h"


using namespace std;

#define PI 3.14159265358979323846
#define N 2048       // 出图的分辨率：N x N
#define M 256        // 接收阵元到发射阵元的最大距离（阵元个数），所以接收孔径为2*M+1
#define ELE_NO 2048  // 发射振元的数量
#define OD 64        // 滤波参数
#define NSAMPLE 3750 // 每一次发射的采样数量

int parallel_emit_sum = 1; // 并行处理多个发射节点，优化使用

/* struct CONST_VALUE
{
    float sample_frequency_div_sound_speed;
    float image_width;
    float image_length;
    float data_diameter;
    int point_length;
    float d_x;
    float d_z;
    int middot; //发射前1us开始接收，也就是约为12.5个点之后发射,数据显示约16个点
                       //const int ELE_NO=1024;
};

__constant__ CONST_VALUE const_value; */

__device__ float dev_ele_coord_x[ELE_NO]; // 写到纹理内存里面
__device__ float dev_ele_coord_y[ELE_NO]; // 写到纹理内存里面
__device__ float dev_filter_data[OD];     // filter parameter

float image_data[N * N] = {0};
int image_point_count[N * N] = {0};

// 原始代码，被 filter_func 取代，但保留以备用
__global__ void kernel3(float *filtered_data, short *data_in_process)
{
    int column_id = blockDim.x * blockIdx.x + threadIdx.x;
    short data[NSAMPLE];
    float filter_temp_data[NSAMPLE];

    if (column_id < gridDim.x * blockDim.x) // 没有意义，但是不能删除
    {
        memset(filter_temp_data, 0, NSAMPLE * sizeof(float));
        for (int sample_cnt = 0; sample_cnt < NSAMPLE; sample_cnt++)
        {
            data[sample_cnt] = data_in_process[sample_cnt * ELE_NO + column_id];
            for (int j = 0; sample_cnt >= j && j < OD; j++)

            {
                filter_temp_data[sample_cnt] += (dev_filter_data[j] * data[sample_cnt - j]);
            }
        }

        for (int i = 0; i < NSAMPLE; i++)
        {
            filtered_data[i * ELE_NO + column_id] = filter_temp_data[i];
        }
    }
}

// 滤波函数
__global__ void filter_func(float *filtered_data, short *data_in_process)
{
    int column_id = blockDim.x * blockIdx.x + threadIdx.x;
    for (int sample_cnt = 0; sample_cnt < NSAMPLE; sample_cnt++)
    {
        for (int j = 0; sample_cnt >= j && j < OD; j++)
        {
            filtered_data[(column_id / 2048) * ELE_NO * NSAMPLE + sample_cnt * ELE_NO + column_id % 2048] += (dev_filter_data[j] * data_in_process[(sample_cnt - j) * ELE_NO + (column_id / 2048) * ELE_NO * NSAMPLE + column_id % 2048]);
        }
    }
}

inline __device__ float distance(float x1, float y1, float x2, float y2) {
    auto dx = x1 - x2;
    auto dy = y1 - y2;
    return sqrtf(dx * dx + dy * dy);
}

bool __device__ __host__ is_close(int delta, int range) {
    int abs_delta = abs(delta);
    return (abs_delta < range || range > 2048 - range);
    // return (delta + range + 2047) % 2048 < 2 * range - 1;
}

__global__ void calc_func(const int global_step, float *image_data,
                          int *point_count, const float *trans_sdata,
                          const int parallel_emit_sum) {
    int sound_speed = 1520;
    float fs = 25e6;
    float image_width = 200.0 / 1000;
    float image_length = 200.0 / 1000;
    float data_diameter = 220.0 / 1000;
    // 3618
    int point_length = data_diameter / sound_speed * fs + 0.5;
    float d_x = image_width / (N - 1);
    float d_z = image_length / (N - 1);
    // magic code...
    int middot =
        -160;    //发射前1us开始接收，也就是约为12.5个点之后发射,数据显示约16个点

    int image_x_id = blockIdx.y;    //线
    int image_z_id = blockIdx.x;    //点
    int image_z_dim = gridDim.x;
    int recv_center_id = threadIdx.x;    //center of 接收阵元

    __shared__ float cache_image[2 * M];
    __shared__ int cache_point[2 * M];
    int cacheIndex = threadIdx.x;

    if (image_x_id < N && image_z_id < N && recv_center_id < 2 * M) {
        float sum_image = 0;
        int sum_point = 0;
        float value_z = -image_length / 2 + d_z * image_z_id;
        float value_x = -image_length / 2 + d_x * image_x_id;
        // what the hell is this !!! need more comments!!!
        float xg = 0.0014;

        for (int step_offset = 0; step_offset < parallel_emit_sum;
             step_offset++) {
            int step = global_step + step_offset;
            int send_id = step;                            // as send_id
            int recv_id = send_id - M + recv_center_id;    //接收阵元
            recv_id = (recv_id + ELE_NO) % ELE_NO;

            float disi = distance(dev_ele_coord_x[send_id],
                                  dev_ele_coord_y[send_id], value_x, value_z);
            float disj = distance(dev_ele_coord_x[recv_id],
                                  dev_ele_coord_y[recv_id], value_x, value_z);
            // what the hell is this !!! need more comments!!!
            // i guess it is a radius?
            float ilength = 112.0 / 1000;
            float imagelength = sqrtf(value_x * value_x + value_z * value_z);
            // 2 * R * disi * cosTheta = R^2 + disi^2 - |(x, z)|^2
            float angle = acosf(
                (ilength * ilength + disi * disi - imagelength * imagelength) /
                2 / ilength / disi);

            // put disi constraint onto for;
            // and since
            auto diff = send_id - recv_id;
            bool is_valid = (disi >= 0.1 * 2 / 3 && is_close(diff, 256)) ||
                            (disi >= 0.1 * 1 / 3 && is_close(diff, 200)) ||
                            (disi >= 0.1 * 0 / 3 && is_close(diff, 100));
            if (is_valid) {
                int num = (disi + disj) / sound_speed * fs + 0.5;

                if (((num + middot + (OD - 1 - 1) / 2) > 100) &&
                    ((num + middot + (OD - 1 - 1) / 2) <= point_length) &&
                    (angle < PI / 9)) {
                    sum_image +=
                        trans_sdata[(num + middot + (OD - 1 - 1) / 2) * ELE_NO +
                                    recv_id + step_offset * ELE_NO * NSAMPLE] *
                        expf(xg * (num - 1));

                    sum_point += 1;
                }
            }
        }
        cache_image[cacheIndex] = sum_image;
        cache_point[cacheIndex] = sum_point;

        __syncthreads();
        // sum up cache_image and cacheIndex, and i have way to make this part disappear
        int step = blockDim.x / 2;
        while (step != 0) {
            if (cacheIndex < step) {
                cache_image[cacheIndex] += cache_image[cacheIndex + step];
                cache_point[cacheIndex] += cache_point[cacheIndex + step];
            }
            __syncthreads();
            step /= 2;
        }

        if (cacheIndex == 0) {
            int pixel_index =
                image_z_id + image_x_id * image_z_dim;    //线程块的索引
            image_data[pixel_index] = cache_image[0];
            point_count[pixel_index] = cache_point[0];
        }
    }
}

__global__ void add(float *sumdata, int *sumpoint, float *imagedata, int *point_count)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N * N)
    {
        sumdata[tid] += imagedata[tid];
        sumpoint[tid] += point_count[tid];
        tid += blockDim.x * gridDim.x;
    }
}

hipError_t precalcWithCuda(short *dev_data_samples_in_process, int ele_emit_id, float *dev_sumdata, int *dev_sumpoint, float *dev_filterdata, float *dev_imagedata, int *dev_pointcount, int parallel_emit_sum)
{
    hipError_t cudaStatus;

    //kernel 1,kernel2 decode
    //kernel3 filter
    hipMemset(dev_filterdata, 0, NSAMPLE * ELE_NO * sizeof(short) * parallel_emit_sum * 2);
    filter_func<<<4 * parallel_emit_sum, 512>>>(dev_filterdata, dev_data_samples_in_process);
    // cudaStatus = cudaGetLastError();
    // if (cudaStatus != cudaSuccess)
    // {
    //     cout << "filter_func launch failed: " << cudaGetErrorString(cudaStatus);
    //     //goto Error;
    //     return cudaStatus;
    // }

    // cudaStatus = cudaDeviceSynchronize();

    dim3 gridimage(N, N);
    //dim3 threads(M);
    calc_func<<<gridimage, 2 * M>>>(ele_emit_id, dev_imagedata, dev_pointcount, dev_filterdata, parallel_emit_sum); //启动一个二维的N*N个block，每个block里面M个thread

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        cout << "calcKernel launch failed: " << hipGetErrorString(cudaStatus);
        //goto Error;
        return cudaStatus;
    }
    // cudaDeviceSynchronize();

    //把所有的结果加到一起
    add<<<32, 32>>>(dev_sumdata, dev_sumpoint, dev_imagedata, dev_pointcount);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        cout << "addKernel launch failed: " << hipGetErrorString(cudaStatus);
        //goto Error;
        return cudaStatus;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    // cudaStatus = cudaDeviceSynchronize();
    // if (cudaStatus != cudaSuccess)
    // {
    //     fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    //     return cudaStatus;
    // }

    return cudaStatus;
}

void get_ele_position(float *ele_coord_x, float *ele_coord_y)
{
    float rfocus = (float)112 / 1000;
    float ele_angle = (2 * PI * 43.4695 / (256 - 1)) / 360; //阵元间隔角度
    float first_one = 2 * PI * (45 - 43.4695) / 360;        //第一个阵元角度

    for (int i = 0; i < 256; i++)
    {
        ele_coord_x[i] = rfocus * cos(first_one + i * ele_angle);
        ele_coord_y[i] = -rfocus * sin(first_one + i * ele_angle);
    }
    for (int i = 256; i < 512; i++)
    {
        ele_coord_x[i] = rfocus * cos(first_one + (i - 256) * ele_angle + PI / 4);
        ele_coord_y[i] = -rfocus * sin(first_one + (i - 256) * ele_angle + PI / 4);
    }
    for (int i = 512; i < 768; i++)
    {
        ele_coord_x[i] = rfocus * cos(first_one + (i - 512) * ele_angle + PI / 2);
        ele_coord_y[i] = -rfocus * sin(first_one + (i - 512) * ele_angle + PI / 2);
    }
    for (int i = 768; i < 1024; i++)
    {
        ele_coord_x[i] = rfocus * cos(first_one + (i - 768) * ele_angle + 3 * PI / 4);
        ele_coord_y[i] = -rfocus * sin(first_one + (i - 768) * ele_angle + 3 * PI / 4);
    }
    for (int i = 1024; i < 1280; i++)
    {
        ele_coord_x[i] = rfocus * cos(first_one + (i - 1024) * ele_angle + PI);
        ele_coord_y[i] = -rfocus * sin(first_one + (i - 1024) * ele_angle + PI);
    }
    for (int i = 1280; i < 1536; i++)
    {
        ele_coord_x[i] = rfocus * cos(first_one + (i - 1280) * ele_angle + 5 * PI / 4);
        ele_coord_y[i] = -rfocus * sin(first_one + (i - 1280) * ele_angle + 5 * PI / 4);
    }
    for (int i = 1536; i < 1792; i++)
    {
        ele_coord_x[i] = rfocus * cos(first_one + (i - 1536) * ele_angle + 3 * PI / 2);
        ele_coord_y[i] = -rfocus * sin(first_one + (i - 1536) * ele_angle + 3 * PI / 2);
    }
    for (int i = 1792; i < 2048; i++)
    {
        ele_coord_x[i] = rfocus * cos(first_one + (i - 1792) * ele_angle + 7 * PI / 4);
        ele_coord_y[i] = -rfocus * sin(first_one + (i - 1792) * ele_angle + 7 * PI / 4);
    }
}

void write_txtfile(std::string output_path)
{
    ofstream outfile(output_path);
    if (!outfile.is_open())
    {
        cout << " the file open fail" << endl;
        exit(1);
    }

    for (int k = 0; k < N; k++)
    {
        for (int j = 0; j < N; j++)
        {
            if (image_point_count[k * N + j] == 0)
                outfile << image_data[k * N + j] << " ";
            else
                outfile << image_data[k * N + j] / image_point_count[k * N + j] << " ";
        }
        outfile << "\r\n";
    }

    outfile.close();
}

int main(int argc, char const *argv[])
{
    time_t start, over;
    start = time(NULL);

    /* float sound_speed = 1520;
    float sample_frequency = 25e6;
    struct CONST_VALUE temp;
    temp.sample_frequency_div_sound_speed = sample_frequency/sound_speed;
    temp.image_width = 200.0 / 1000;
    temp.image_length = 200.0 / 1000;
    temp.data_diameter = 220.0 / 1000;
    temp.point_length = temp.data_diameter * temp.sample_frequency_div_sound_speed + 0.5;
    temp.d_x = temp.image_width / (N - 1);
    temp.d_z = temp.image_length / (N - 1);
    temp.middot = -160; //发射前1us开始接收，也就是约为12.5个点之后发射,数据显示约16个点
                       //const int ELE_NO=1024;

    if (cudaMemcpyToSymbol(&const_value, &temp, sizeof(temp)) != cudaSuccess)
    {
        cout << "ERROR :: struct CONST_VALUE copy failed." << endl;
    } */

    std::string filter_path = "";
    std::string bin_path = "";
    std::string output_path = "";
    switch (argc)
    {
    case 4:
        parallel_emit_sum = atoi(argv[1]);
        filter_path = argv[2];
        bin_path = argv[3];
        output_path = "origin.txt";
        break;
    case 5:
        parallel_emit_sum = atoi(argv[1]);
        filter_path = argv[2];
        bin_path = argv[3];
        output_path = argv[4];
        break;
    default:
        std::cout << "Please input 3 or 4 paras" << std::endl;
        std::cout << "[parallel emit sum] [filter path] [bin path]" << std::endl;
        std::cout << "[parallel emit sum] [filter path] [bin path] [output path]" << std::endl;
        exit(-1);
        break;
    }

    hipError_t cudaStatus;

    time_t start_read, over_read;
    start_read = time(NULL);
    // Read filter data and put in GPU
    ifstream file_read;
    file_read.open(filter_path.c_str(), ios_base::in | ios::binary);
    if (!file_read.is_open())
    {
        cout << " the file filter open fail" << endl;
        return -1;
    }
    float filter_data[OD];
    for (int ii = 0; ii < OD; ii++)
    {
        file_read.read((char *)&filter_data[ii], sizeof(float));
    }
    file_read.close();
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(dev_filter_data), filter_data, sizeof(float) * OD);

    if (cudaStatus != hipSuccess)
    {
        cout << "center Fail to hipMemcpyToSymbol on GPU" << endl;
        return;
    }

    file_read.open(bin_path.c_str(), ios_base::in | ios::binary | ios::ate);
    if (!file_read.is_open())
    {
        cout << " the bin file open fail" << endl;
        return -1;
    }
    long long int filesize = file_read.tellg();
    file_read.seekg(0, file_read.beg);
    // 为 bin_buffer 申请空间，并把 filepath 的数据载入内存
    char *bin_buffer = (char *)std::malloc(filesize);
    if (bin_buffer == NULL)
    {
        std::cout << "ERROR :: Malloc data for buffer failed." << std::endl;
        return 0;
    }
    file_read.read(bin_buffer, filesize);
    if (file_read.peek() == EOF)
    {
        file_read.close();
    }
    else
    {
        std::cout << "ERROR :: Read bin file error." << std::endl;
        file_read.close();
        // exit(-1);
    }
    over_read = time(NULL);
    cout << "Reading time is : " << difftime(over_read, start_read) << "s!" << endl;

    //image grid
    float ele_coord_x[ELE_NO] = {0};
    float ele_coord_y[ELE_NO] = {0};
    get_ele_position(&ele_coord_x[0], &ele_coord_y[0]);

    if (hipMemcpyToSymbol(HIP_SYMBOL(dev_ele_coord_x), ele_coord_x, sizeof(float) * ELE_NO) != hipSuccess)
    {
        cout << "ERROR :: Failed for hipMemcpyToSymbol dev_ele_coord_x." << endl;
        return -1;
    }

    if (hipMemcpyToSymbol(HIP_SYMBOL(dev_ele_coord_y), ele_coord_y, sizeof(float) * ELE_NO) != hipSuccess)
    {
        cout << "ERROR :: Failed for hipMemcpyToSymbol dev_ele_coord_y." << endl;
        return -1;
    }

    float *dev_sumdata;
    int *dev_sumpoint;
    if (hipMalloc((void **)(&dev_sumdata), N * N * sizeof(float)) != hipSuccess)
    {
        cout << "ERROR :: Failed for hipMalloc dev_sumdata." << endl;
        return -1;
    }
    if (hipMalloc((void **)(&dev_sumpoint), N * N * sizeof(int)) != hipSuccess)
    {
        cout << "ERROR :: Failed for hipMalloc dev_sumpoint." << endl;
        return -1;
    }
    // init dev_sumdata and dev_sumpoint
    if (hipMemcpy(dev_sumdata, image_data, N * N * sizeof(float), hipMemcpyHostToDevice) != hipSuccess)
    {
        cout << "ERROR :: Failed for hipMemcpy dev_sumdata." << endl;
        return -1;
    }
    if (hipMemcpy(dev_sumpoint, image_point_count, N * N * sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
    {
        cout << "ERROR :: Failed for hipMemcpy dev_sumpoint." << endl;
        return -1;
    }

    long long length_of_data_in_process = NSAMPLE * ELE_NO * sizeof(short) * parallel_emit_sum;
    short *dev_data_samples_in_process;
    float *dev_filterdata;

    cudaStatus = hipMalloc((void **)(&dev_data_samples_in_process), length_of_data_in_process);
    if (cudaStatus != hipSuccess)
    {
        cout << "data_samples_in_process Fail to hipMalloc on GPU" << endl;
        return -1;
    }

    if (hipMalloc((void **)(&dev_filterdata), length_of_data_in_process * 2) != hipSuccess) // 转 float 乘以 2
    {
        cout << "ERROR :: Failed for hipMalloc dev_filterdata." << endl;
        return -1;
    }

    float *dev_imagedata;
    // float *dev_trans_sdata;

    int *dev_pointcount;
    //  int dev_i;

    cudaStatus = hipMalloc((void **)(&dev_imagedata), N * N * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        cout << "imagedata Fail to hipMalloc on GPU" << endl;
        //goto Error;
        return cudaStatus;
    }
    cudaStatus = hipMalloc((void **)(&dev_pointcount), N * N * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        cout << "pointcount Fail to hipMalloc on GPU" << endl;
        //goto Error;
        return cudaStatus;
    }

    long long bin_buffer_index = 0;
    for (int ele_emit_id = 0; ele_emit_id < ELE_NO; ele_emit_id += parallel_emit_sum)
    //for (i=1;i<=1;i++)
    {
        printf("Number of element : %d\n", ele_emit_id);

        // memcpy(&data_samples_in_process[0], &bin_buffer[bin_buffer_index], length_of_data_in_process);
        // bin_buffer_index = bin_buffer_index + length_of_data_in_process;

        // cudaStatus = cudaMemcpy(dev_data_samples_in_process, data_samples_in_process, length_of_data_in_process, cudaMemcpyHostToDevice);
        cudaStatus = hipMemcpy(dev_data_samples_in_process, &bin_buffer[bin_buffer_index], length_of_data_in_process, hipMemcpyHostToDevice);
        bin_buffer_index = bin_buffer_index + length_of_data_in_process;
        if (cudaStatus != hipSuccess)
        {
            cout << "data_samples_in_process Fail to hipMemcpy on GPU" << endl;
            //goto Error;
            return cudaStatus;
        }
        cudaStatus = precalcWithCuda(dev_data_samples_in_process, ele_emit_id, dev_sumdata, dev_sumpoint, dev_filterdata, dev_imagedata, dev_pointcount, parallel_emit_sum);
        //}
        // over=time(NULL);
        // cout<<"Running time is : "<<difftime(over,start)<<"s!"<<endl;
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "calcWithCuda failed!");
            return 1;
        }
        // cudaError_t cudaStatus = calcWithCuda( i,dev_sumdata,dev_sumpoint,dev_filterdata);
    }
    cudaStatus = hipMemcpy(image_data, dev_sumdata, N * N * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        cout << "allimagedata Fail to hipMemcpy to CPU" << endl;
        return 1;
        //goto Error;
    }

    cudaStatus = hipMemcpy(image_point_count, dev_sumpoint, N * N * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        cout << "allpointcount Fail to hipMemcpy to CPU" << endl;
        return 1;
        //goto Error;
    }

    write_txtfile(output_path);
    over = time(NULL);
    cout << "Running time is : " << difftime(over, start) / 60 << "min!" << endl;
    hipFree(dev_sumdata);
    hipFree(dev_sumpoint);
    hipFree(dev_data_samples_in_process);
    hipFree(dev_filterdata);
    hipFree(dev_imagedata);
    hipFree(dev_pointcount);
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
}

//cudaStatus= cudaMemcpy( data_in_process,dev_filterdata , 5000*1024 * sizeof(float),cudaMemcpyDeviceToHost ) ;
// if (cudaStatus != cudaSuccess) {
//cout<<"data_output Fail to cudaMemcpy to CPU"<<endl;
// goto Error;
//goto Error;
// }
